#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <string>
#include <functional>
#include <initializer_list>
#include <algorithm>
#include "TensorBlock.cuh"
#include <jsoncpp/json/json.h>
#include <omp.h>
#include <filesystem>

inline void StoreVector(std::basic_fstream<char>&& FileObjs){
    uint32_t lvalue = 10;
    std::cout << &lvalue << " Stack Memory Addr " << std::endl;
    std::vector<float3> VectorStorage[4];
    Json::Reader _ReadJson;
    Json::Value _ValueReader;
    _ReadJson.parse(FileObjs,_ValueReader);
    std::cout << _ValueReader.operator[]("result").operator[]("records").size() << " Size of Array element in vector " << std::endl;
    
    omp_set_num_threads(4);
    std::vector<std::array<float3,4>> StorageVector;
    StorageVector.reserve(1000);
    uint16_t VectorSize[4][16];
    VectorSize[0][0] = 1;
    VectorSize[1][0] = 201;
    VectorSize[2][0] = 401;
    VectorSize[3][0] = 601;
    uint16_t WorkCycle = 0;
    std::cout << _ValueReader.operator[]("result").operator[]("records").operator[](VectorSize[3][0]).operator[]("comp_sora_1m").asFloat() << " Floating point result " << std::endl;

    #pragma omp parallel
    {
        int ThreadIds = omp_get_thread_num();
        for (;WorkCycle < 200;WorkCycle++){
            StorageVector.operator[](VectorSize[ThreadIds][0]).operator[](0).x = _ValueReader.operator[]("result").operator[]("records").operator[](VectorSize[ThreadIds][0]).operator[]("comp_sora_1m").asFloat();
            StorageVector.operator[](VectorSize[ThreadIds][0]).operator[](0).y = _ValueReader.operator[]("result").operator[]("records").operator[](VectorSize[ThreadIds][0]).operator[]("comp_sora_3m").asFloat();
            StorageVector.operator[](VectorSize[ThreadIds][0]).operator[](0).z = _ValueReader.operator[]("result").operator[]("records").operator[](VectorSize[ThreadIds][0]).operator[]("comp_sora_6m").asFloat();
            VectorSize[ThreadIds][0] = VectorSize[ThreadIds][0] + 1;
        }
    }
    return;
}

inline void StrConvertFlt(const char* strliteral){
    float retval        {0.0};
    int Beforevalue = -1;
    const int len = (int)strlen(strliteral);
    int i = 0;
    int j = 0;
    while ((int)*(strliteral+i)  != 46){
        Beforevalue++;
        i++;
        continue;
    }
    i = 0;
    for (;j < len;){
        switch (i = (int)*strliteral++){
            case 48 ... 57:{
                retval = retval + ((float)(i - 0b00110000) * (pow(10.0f,(float)Beforevalue)));  //  expensive ops
                Beforevalue--;
                break;
            }
            case 46:{
                Beforevalue = -1; // turn-on switch
                break;
            }
            default:{break;}
        }
        j++;
        continue;
    }
    //std::cout << "Conversion Results " << retval << " || " << retval * 2.52f << std::endl;
    return;
}

// Overload function
inline void StrConvertFlt(const std::tuple<std::string,std::string>& strLiteral, std::tuple<std::array<float,16>,std::array<uint32_t,2>,uint32_t>& StorageArr){
    float retval        {0.0};
    int Beforevalue = -1;
    const int len = (int)(std::get<0>(strLiteral).length());
    int n_stackvar = -0;
    int i = 0;
    int j = 0;
    while ((int)(std::get<0>(strLiteral).operator[](i))  != 46){
        Beforevalue++;
        i++; // Expression statement; ending with semi-colon
        continue;
    }
    i = 0;
    for (;j < len;){
        switch (i = ((int)(std::get<0>(strLiteral).operator[](n_stackvar++)))){
            case 48 ... 57:{
                retval = retval + ((float)(i - 0b00110000) * (pow(10.0f,(float)Beforevalue)));  // most expensive operations
                Beforevalue--;
                break;
            }
            case 46:{
                Beforevalue = -1; // turn-on switch
                break;
            }
            default:{break;}
        }
        j++;
        std::get<0>(StorageArr).operator[](0) = retval;
        continue;
    }
    //std::cout << "Conversion Results " << retval << " Stored Results " << std::get<0>(StorageArr).operator[](0) << std::endl;
    return;
}


void CvrtStrToInt(const char* strLite){
    std::size_t xvalue = strlen(strLite);
    uint16_t Powvalue  {0};
    uint32_t AddStore {0};
    uint32_t Basevalue {0};
    uint16_t i {1};
    while (i < xvalue+1){
        switch ((int)strLite[xvalue-i]){
            case 44:{__asm__("nop");break;} 
            default:{
                AddStore = AddStore + ((uint32_t)strLite[xvalue-i] - 0b00110000) * (uint32_t)(pow(10.0f,(float)Basevalue));
                Basevalue = Basevalue + 1;
                break;
            }
        }
        i++;
        continue;
    }
    return;
}

template<typename T,std::size_t XGET>
void CvrtStrToInt(const char* strLite,std::tuple<std::array<float,16>,std::array<uint32_t,2>,uint32_t>& refArg){
    std::size_t xvalue = strlen(strLite);
    uint16_t Powvalue  {0};
    T AddStore {0};
    T Basevalue {0};
    uint16_t i {1};
    while (i < xvalue+1){
        switch ((int)strLite[xvalue-i]){
            case 44:{__asm__("nop");break;} 
            default:{
                AddStore = AddStore + ((uint32_t)strLite[xvalue-i] - 0b00110000) * (uint32_t)(pow(10.0f,(float)Basevalue)); // conversion to INT
                Basevalue = Basevalue + 1;
                break;
            }
        }
        i++;
        continue;
    }
    std::get<XGET>(refArg)[0] = AddStore;
    return;
}


uint32_t xvalues = 0;
template<std::size_t Z,std::size_t X>
void OperateData(std::tuple<std::string,std::string>& TupleArr,std::string& refArg,int16_t t,const std::vector<std::tuple<uint32_t,std::size_t,uint16_t,uint16_t>>& CommaPos){
    uint32_t y = 2;
    for (;(int)refArg.operator[](std::get<0>(CommaPos.operator[](xvalues+X))+y)!=34; y=y+1){
        std::get<X>(TupleArr).operator+=(refArg.operator[](std::get<0>(CommaPos.operator[](xvalues+X))+y)); // operator += to keep accumulating
    }
    xvalues = xvalues + Z;
    return;
}

std::tuple<std::string,std::string>* ParseCSV(std::string& refArg){
    std::string::iterator iterobjs = refArg.begin();
    std::vector<uint32_t> NewlinePos{};
    std::vector<std::tuple<uint32_t,std::size_t,uint16_t,uint16_t>> CommaPos{};
    std::vector<std::tuple<uint16_t,uint16_t>> DataPos{};
    uint16_t Newline {0};
    int32_t CommaCount {0};
    uint32_t WordCounter {0};
    uint32_t singleLinePos = 0;
    while (iterobjs.operator*() != '\0'){
        switch ((int)refArg.operator[](WordCounter)){
            case 10:{NewlinePos.push_back(WordCounter);singleLinePos=0;CommaCount=0;Newline++;break;}
            case 44:{CommaCount++;CommaPos.push_back({WordCounter,singleLinePos,Newline,CommaCount});DataPos.push_back({Newline,CommaCount});singleLinePos++;break;}
            default:{singleLinePos++;break;}
        }
        WordCounter++;
        iterobjs++;
        continue;
    }
    uint32_t InnerLoop = 0;
   //std::tuple<std::string,std::string> TupleArr[Newline+1]{};
   std::tuple<std::string,std::string>* TupleArr{ (std::tuple<std::string,std::string>* )std::malloc(10*9700)}; // an array of empty tuples 
    for (uint16_t t = 0;t <= Newline;){
        uint32_t Tracker = 1;
        if (std::get<0>(DataPos.operator[](InnerLoop+1)) == std::get<0>(DataPos.operator[](InnerLoop))){
            for (;std::get<0>(DataPos.operator[](InnerLoop+1)) == std::get<0>(DataPos.operator[](InnerLoop));){
                Tracker++;  // Tracker find the difference between current comma and the next one
                InnerLoop++;
                continue;
            }
            switch (Tracker){
                case 2:{if (t==0){xvalues = xvalues + Tracker;break;}OperateData<2,0>(TupleArr[t],refArg,t,CommaPos);break;}
                case 4:{if (t==0){xvalues = xvalues + Tracker;break;}OperateData<4,0>(TupleArr[t],refArg,t,CommaPos);break;}
                case 5:{if (t==0){xvalues = xvalues + Tracker;break;}OperateData<5,0>(TupleArr[t],refArg,t,CommaPos);break;}
                default:{OperateData<3,0>(TupleArr[t],refArg,t,CommaPos);break;}
            }
        }
        t++;
        InnerLoop++;
        continue;
    }
    xvalues = 0;
    InnerLoop = 0;
    for (uint16_t t = 0;t <= Newline;){
        uint32_t Tracker = 1;
        if (std::get<0>(DataPos.operator[](InnerLoop+1)) == std::get<0>(DataPos.operator[](InnerLoop))){
            for (;std::get<0>(DataPos.operator[](InnerLoop+1)) == std::get<0>(DataPos.operator[](InnerLoop));){
                Tracker++;  // Tracker find the difference between current comma and the next comma
                InnerLoop++;
                continue;
            }
            switch (Tracker){
                case 2:{if (t==0){xvalues = xvalues + Tracker;break;}OperateData<2,1>(TupleArr[t],refArg,t,CommaPos);break;}
                case 4:{if (t==0){xvalues = xvalues + Tracker;break;}OperateData<4,1>(TupleArr[t],refArg,t,CommaPos);break;}
                case 5:{if (t==0){xvalues = xvalues + Tracker;break;}OperateData<5,1>(TupleArr[t],refArg,t,CommaPos);break;}
                default:{OperateData<3,1>(TupleArr[t],refArg,t,CommaPos);break;}
            }
        }
        t++;
        InnerLoop++;
        continue;
    }

    return TupleArr;
}

int main(int args,const char** argvec){
    CvrtStrToInt("18,192,500");
    std::basic_fstream<char> fileInfos{"/home/cyang279/TensorsCapital/QCOM-07082023_intraday.csv",std::ios::in};
    float y = 13.5435;
    std::cout << log(y) << " Log values of y " << std::endl;
    long FileBegin {fileInfos.tellg()};
    fileInfos.seekg(0,std::ios::end); // function call expression -- statement with semicolon
    long FileEnd {fileInfos.tellg()};
    fileInfos.seekg(0,std::ios::beg);
    const std::size_t FileSize = (std::size_t)(FileEnd - FileBegin);
    std::string BufString{};
    BufString.reserve(FileSize);
    fileInfos.read(&BufString.operator[](0),FileSize);
    std::tuple<std::string, std::string>* TupsData = ParseCSV(BufString);
    std::cout << " Reading call return value " << std::endl;
    // Converting string to float and uint32
    std::tuple<std::array<float,16>,std::array<uint32_t,2>,uint32_t> ConvertedData[392];
    std::cout << sizeof(ConvertedData) << " Size of this element " << std::endl;
    
    int DataPos[3][16];         //
    DataPos[0][0] = 1;
    DataPos[1][0] = 108;
    DataPos[2][0] = 215;

    std::cout << " ************************** " << std::endl;
    omp_set_num_threads(3);
    #pragma omp parallel
    {
        int Icount = 0;
        int ids = omp_get_thread_num();
        for (;Icount < 43;){
            StrConvertFlt(TupsData[DataPos[ids][0]],ConvertedData[DataPos[ids][0]]);
            DataPos[ids][0] = (DataPos[ids][0])+1;
            Icount = Icount + 1;
            continue;
        }
    }
    // reset the variables
    DataPos[0][0] = 1;
    DataPos[1][0] = 108;
    DataPos[2][0] = 215;
    #pragma omp parallel
    {
        int Icount = 0;
        int ids = omp_get_thread_num();
        for (;Icount < 43;){
            CvrtStrToInt<uint32_t,1>(std::get<1>(TupsData[DataPos[ids][0]]).c_str(),ConvertedData[DataPos[ids][0]]);
            DataPos[ids][0] = (DataPos[ids][0])+1;
            Icount = Icount + 1;
            continue;
        }
    }

    // tuple is unordered map
    float* BaseAddr {(float*)&std::get<0>(ConvertedData[1]).operator[](0)};
    
    std::cout << " Expression should return a boolean " << (*BaseAddr == *(BaseAddr + 19)) << std::endl;
    std::cout << *BaseAddr << " || " << *(BaseAddr+19) << " || " << *(BaseAddr+57) << std::endl;
    std::qsort((void*)BaseAddr,50,76,(int (*)(const void*,const void*))&FuncCmpFlt);
    std::cout << *BaseAddr << " || " << *(BaseAddr+19) << " || " << *(BaseAddr+57) << std::endl;


    uint32_t Dim2Arr[][3] {{58,7,32},
                            {75,58,95},
                            {16,23,45},
                            {68,758,96},
                            {69,79,96}
                            };

    std::cout << *(*(Dim2Arr)+1) << " Before sort " << std::endl;
    std::cout << *(*(Dim2Arr+3)+1) << " Before sort " << std::endl;

    //std::qsort((void*)Dim2Arr,12,4,(int (*)(const void*,const void*))&FuncCompare);



    std::cout << " ^^^^^ " << std::get<1>(ConvertedData[95])[0] << " ****** " << std::get<1>(ConvertedData[25])[0] << std::endl;
    StoreVector(std::basic_fstream<char>{"/home/cyang279/TensorsCapital/ExtendedSora.json"});
    std::vector<float3> CudaObjs {{0.34,15.33,17.39},{17.657,89.325,48.652},{27.548,72.315,68.755}};
    std::string Csvrawvalue {"15.34"};
    std::cout << strtof(Csvrawvalue.c_str(),nullptr) << " Result from conversion " << std::endl;
    std::vector<float3>::iterator IterObjs = CudaObjs.begin();
    operator<<(std::cout.operator<<(IterObjs->x), " Floating Point ").operator<<(std::endl);
    auto LambdaExpr = [](std::string& RefObjs)->int{
        std::cout << RefObjs << " Memory Addrs " << std::endl;
        (void)std::system(RefObjs.c_str());
        return (int)RefObjs.length();
    };
    std::string Cmdlines {"pwd"};
    LambdaExpr(Cmdlines);
    return EXIT_SUCCESS;
}